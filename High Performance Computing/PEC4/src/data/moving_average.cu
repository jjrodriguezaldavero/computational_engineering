
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <iostream>
#include <fstream>

using namespace std;

//CUDA kernels
__global__ void smooth(float *in, float *out, int n) {
	//Get our global and block thread ID
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	if (col < n) {
		float tmpAvg = 0.0;
		for (int i = 0; i < 9; i++) {
			if (col - i >= 0) {
				tmpAvg += *(in + col-i) ;
			} else {
				tmpAvg += 0;
			}
		}
		
		//out[col][row] = tmpAvg;
		*(out + col) = tmpAvg / (float)9.0;
	}
}
	
//Main function	
int main(int argc, char *argv[]) {
	//Length of array
	//Matriz 1000*50
	int n = 50000;
	
	//Device input array
	float *d_A;
	
	//Device output array
	float *d_B;
	
	//Host input array
	float *h_A;
	
	//Host output arra
	float *h_B;
	
	//Size, in bytes, of each array
	size_t bytes = n*sizeof(float);
	
	//Allocate host arrays
	h_A = (float*)malloc(bytes);
	h_B = (float*)malloc(bytes);
	
	//Allocate device arrays
	hipMalloc(&d_A, bytes);
	hipMalloc(&d_B, bytes);
	
	//Initialize content of input array
	int i;
	for(i=0; i<n; i++) {
		*(h_A + i) = rand() % 10 + 1;
	}

	//Copy host vector to device
	hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice);
	
	//Number of threads in each thread block
	dim3 blockSize(32,1,1);
	
	//Number of thread blocks in grid
	int grid = (int)ceil((float)n / blockSize.x);
	dim3 gridSize(grid,1,1);
	
	//Execute the kernel
	smooth<<<gridSize,blockSize>>>(d_A, d_B, n);
	
	//Copy device vector to host
	hipMemcpy(h_B, d_B, bytes, hipMemcpyDeviceToHost);
	
	//Write array to txt file
	ofstream myfile ("moving_average.txt");
	if (myfile.is_open())
	{
		for(int i = 0; i < n; i ++){
			myfile << *(h_B + i) << " " ;
		}
		myfile.close();
	} else cout << "Unable to open file";
	
	//Release device memory
	hipFree(d_A);
	hipFree(d_B);

	//Release host memory
	free(h_A);
	free(h_B);
	
	return 0;
	}