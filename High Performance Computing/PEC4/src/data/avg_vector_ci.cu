
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <iostream>
#include <fstream>

using namespace std;

//CUDA kernels
__global__ void avg(float *in, float *out, int n, int m) {
	//Identify the thread ID as the column number
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	//Calculate the loop iterations for the corresponding vector elements
	if (col < n) {
		float tmpAvg = 0.0;
		//Calculate the sum of all the elements in the column
		for (int i = 0; i < m; i++) {
			int k = n * i;
			tmpAvg += *(in + col + k) ;
		}
		*(out + col) = tmpAvg;
	}
}
	
//Main function	
int main(int argc, char *argv[]) {
	
	//Define matrix dimensions and pointers
	int n = 1000; //columns
	int m = 50; //rows 
	float *d_A;
	float *d_B;
	float *h_A;
	float *h_B;
	
	size_t bytes = n*m*sizeof(float);
	size_t bytes_vec = n*sizeof(float);
	
	//Allocate host arrays
	h_A = (float*)malloc(bytes);
	h_B = (float*)malloc(bytes_vec);
	
	//Allocate device arrays
	hipMalloc(&d_A, bytes);
	hipMalloc(&d_B, bytes_vec);
	
	//Initialize content of input array
	int i;
	ifstream file("file.txt");
    if(file.is_open())
    {
        for(i=0; i<n; i++) {
            file >> *(h_A + i);
        }
    }

	//Copy host vector to device
	hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice);
	
	//Define grid dimensions
	dim3 blockSize(500,1,1);
	int grid = (int)ceil((float)(n*m) / blockSize.x);
	dim3 gridSize(grid,1,1);
	
	//Execute the kernel
	avg<<<gridSize,blockSize>>>(d_A, d_B, n, m);
	
	//Copy device vector to host
	hipMemcpy(h_B, d_B, bytes_vec, hipMemcpyDeviceToHost);
	
	//Write array to txt file
	ofstream myfile ("avg_vector.txt");
	if (myfile.is_open())
	{
		for(int i = 0; i < n; i ++){
			myfile << *(h_B + i) << " " ;
		}
		myfile.close();
	} else cout << "Unable to open file";
	
	//Release device and host memory
	hipFree(d_A);
	hipFree(d_B);
	free(h_A);
	free(h_B);
	
	return 0;
	}